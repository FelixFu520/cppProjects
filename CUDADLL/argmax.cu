#include "hip/hip_runtime.h"
#include "argmax.cuh"

/* ---------------- �������(����Demo) ----------------*/
// �˺���
__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
// �������
int myVectorAdd(int* a, int* b, int* c, int size)
{
	int result = -1;
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// ѡ���������е�GPU  
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		result = 1;
		goto Error;
	}

	// ��GPU��Ϊ����dev_a��dev_b��dev_c�����ڴ�ռ�.  
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		result = 2;
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		result = 3;
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		result = 4;
		goto Error;
	}

	// �������ڴ渴�����ݵ�GPU�ڴ���.  
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		result = 5;
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		result = 6;
		goto Error;
	}

	// ����GPU�ں˺���  
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// ����hipDeviceSynchronize�ȴ�GPU�ں˺���ִ����ɲ��ҷ����������κδ�����Ϣ  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		result = 7;
		goto Error;
	}

	// ��GPU�ڴ��и������ݵ������ڴ���  
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		result = 8;
		goto Error;
	}

	result = 0;

	// ����CUDA�豸�����˳�֮ǰ�������hipDeviceReset  
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 9;
	}
Error:
	//�ͷ��豸�б�����ռ�ڴ�  
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return result;
}

/* ----------------  ��CHW��Cά���ϵ����ֵ����λ�� ----------------*/
// �˺���
__global__ void compareMaxValue(float* inputImages, unsigned char* outputMaxValues, unsigned char* outputMaxIndices, int width, int height, int numImages)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < width * height)
	{
		float maxValue = -FLT_MAX;
		unsigned char maxIndex = 0;

		for (int i = 0; i < numImages; i++)
		{
			float value = inputImages[i * width * height + index];
			if (value > maxValue)
			{
				maxValue = value;
				maxIndex = i;
			}
		}
		
		outputMaxIndices[index] = maxIndex;
		if (maxIndex == 0) {
			outputMaxValues[index] = 0;
		}
		else {
			outputMaxValues[index] = maxValue*255;
		}
	}
}
__global__ void compareMaxValue(float* inputImages, float* outputMaxValues, float* outputMaxIndices, int width, int height, int numImages)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < width * height)
	{
		float maxValue = -FLT_MAX;
		unsigned char maxIndex = 0;

		for (int i = 0; i < numImages; i++)
		{
			float value = inputImages[i * width * height + index];
			if (value > maxValue)
			{
				maxValue = value;
				maxIndex = i;
			}
		}

		outputMaxIndices[index] = maxIndex;
		if (maxIndex == 0) {
			outputMaxValues[index] = 0;
		}
		else {
			outputMaxValues[index] = maxValue * 255;
		}
	}
}
__global__ void compareMaxValue(float* inputImages, unsigned char* outputMaxValues, unsigned char* outputMaxIndices, int width, int height, int channels, int numImages)
{
	//int index = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;  // ÿ���̴߳���һ�����ص�
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < width * height * numImages)
	{
		float maxValue = -FLT_MAX;
		unsigned char maxIndex = 0;

		for (int i = 0; i < channels; i++)
		{
			float value = inputImages[i * width * height * numImages + index];
			if (value > maxValue)
			{
				maxValue = value;
				maxIndex = i;
			}
		}

		outputMaxIndices[index] = maxIndex;
		if (maxIndex == 0) {
			outputMaxValues[index] = 0;
		}
		else {
			outputMaxValues[index] = (unsigned char)(maxValue * 255);
		}
	}
}
// �����ֵ����λ��
int argmaxChannels(float* inputImages, unsigned char* outputMaxValues, unsigned char* outputMaxIndices, int width, int height, int numImages) {
	// Allocate GPU memory for input and output
	float* d_inputImages;
	unsigned char* d_outputMaxValues;
	unsigned char* d_outputMaxIndices;
	hipMalloc((void**)&d_inputImages, numImages * width * height * sizeof(float));
	hipMalloc((void**)&d_outputMaxValues, width * height * sizeof(unsigned char));
	hipMalloc((void**)&d_outputMaxIndices, width * height * sizeof(unsigned char));

	// Copy input data from host to device
	hipMemcpy(d_inputImages, inputImages, numImages * width * height * sizeof(float), hipMemcpyHostToDevice);

	// Invoke the CUDA kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
	compareMaxValue << <blocksPerGrid, threadsPerBlock >> > (d_inputImages, d_outputMaxValues, d_outputMaxIndices, width, height, numImages);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Copy output data from device to host
	hipMemcpy(outputMaxValues, d_outputMaxValues, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(outputMaxIndices, d_outputMaxIndices, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

	// Free GPU memory
	hipFree(d_inputImages);
	hipFree(d_outputMaxValues);
	hipFree(d_outputMaxIndices);

	return 0;
}
int argmaxChannels(const std::vector<std::vector<cv::Mat>>& inputImages, std::vector<cv::Mat>& outputMaxValues, std::vector<cv::Mat>& outputMaxIndices, int width, int height, int channels, int numImages) {
	// Allocate GPU memory for input and output
	float* d_inputImages;
	unsigned char* d_outputMaxValues;
	unsigned char* d_outputMaxIndices;
	hipMalloc((void**)&d_inputImages, numImages * channels * width * height * sizeof(float));
	hipMalloc((void**)&d_outputMaxValues, numImages * width * height * sizeof(unsigned char));
	hipMalloc((void**)&d_outputMaxIndices, numImages * width * height * sizeof(unsigned char));

	// Copy input data from host to device
	for (int c = 0; c < channels; c++) {
		for (int n = 0; n < numImages; n++) {
			hipMemcpy(d_inputImages + c*n*height*width + n*height*width, inputImages[n][c].data, width * height * sizeof(float), hipMemcpyHostToDevice);
		}
	}


	// Invoke the CUDA kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
	compareMaxValue << <blocksPerGrid, threadsPerBlock >> > (d_inputImages, d_outputMaxValues, d_outputMaxIndices, width, height, channels, numImages);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Copy output data from device to host
	for (int n = 0; n < numImages; n++) {
		hipMemcpy(outputMaxValues[n].data, d_outputMaxValues + n * width * height, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipMemcpy(outputMaxIndices[n].data, d_outputMaxIndices + n * width * height, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
	}


	// Free GPU memory
	hipFree(d_inputImages);
	hipFree(d_outputMaxValues);
	hipFree(d_outputMaxIndices);

	return 0;
}
int argmaxChannels(const std::vector<cv::cuda::GpuMat>& inputImages, cv::cuda::GpuMat& outputMaxValues, cv::cuda::GpuMat& outputMaxIndices, int width, int height, int numImages) {

	float* d_inputImages;
	hipMalloc((void**)&d_inputImages, numImages * width * height * sizeof(float));

	for (int n = 0; n < inputImages.size(); n++) {
		hipMemcpy(
			d_inputImages + n * height * width,
			inputImages[n].data,
			height * width * sizeof(float), 
			hipMemcpyDeviceToDevice
		);
	}

	// Invoke the CUDA kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
	compareMaxValue << <blocksPerGrid, threadsPerBlock >> > (d_inputImages, (float*)outputMaxValues.data, (float*)outputMaxIndices.data, width, height, numImages);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Free GPU memory
	hipFree(d_inputImages);

	return 0;
}